#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 * Copyright (c) 2021, Max von Buelow, GRIS, Technical University of Darmstadt
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *	notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *	notice, this list of conditions and the following disclaimer in the
 *	documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *	contributors may be used to endorse or promote products derived
 *	from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <string>
#include <map>
#include <unordered_set>
#include <cstring>
#include <fstream>
#include <iomanip>
#include <limits>
#include <set>
#include <stack>
#include <sstream>
#include <stdexcept>
#include <random>
#include <algorithm>
#include <list>
#include <numeric>
#include <random>
#include "meminf_storage.h"
#include "cachesim.h"
#include "plugin.h"
#include "coalescing.h"

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the mem_access_t structure */
#include "hiacc.h"

#include <stdlib.h>
#include <dlfcn.h>


/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 20)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;

/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;
std::map<int, memop> memops;


#define GETPRED(m, i) ((m >> (31 - i)) & 1)


std::list<void*> plugins;
CacheSim cachesim;

void nvbit_at_init() {
	setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
	GET_VAR_INT(
		instr_begin_interval, "INSTR_BEGIN", 0,
		"Beginning of the instruction interval where to apply instrumentation");
	GET_VAR_INT(
		instr_end_interval, "INSTR_END", UINT32_MAX,
		"End of the instruction interval where to apply instrumentation");
	GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
	std::string pad(100, '-');
	printf("%s\n", pad.c_str());

	std::string pluginpaths;
	GET_VAR_STR(pluginpaths, "PLUGINS", "Plugins as .so files, comma separated");
	std::stringstream sstream(pluginpaths);
	std::string plugin;
	while (std::getline(sstream, plugin, ',')){
		std::cout << "Loading plugin " << plugin << std::endl;
		void *handle = dlopen(plugin.c_str(), RTLD_LAZY);
		plugins.push_back(handle);
	}

	GET_VAR_INT(cachesim.cacheconf.l1_capacity, "L1SIZE", 57 * 1024, "Capacity of L1 cache in bytes");
	GET_VAR_INT(cachesim.cacheconf.l2_capacity, "L2SIZE", 11/*11*/ /* 5.5, see 2 on RHS */ * 1024 * 1024 / 2, "Capacity of L2 cache in bytes");
	GET_VAR_INT(cachesim.cacheconf.l2_assoc, "L2ASSOC", 16, "Associativity of L2 cache");

}
/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;


memop parse_op(Instr &instr)
{
	memop res;
	res.ld = instr.isLoad();
	res.st = instr.isStore();
	memop::MemorySpace map[128];
	map[(int)InstrType::MemorySpace::LOCAL] = memop::LOCAL;
	map[(int)InstrType::MemorySpace::GENERIC] = memop::GENERIC;
	map[(int)InstrType::MemorySpace::GLOBAL] = memop::GLOBAL;
	map[(int)InstrType::MemorySpace::SHARED] = memop::SHARED;
	map[(int)InstrType::MemorySpace::GLOBAL_TO_SHARED] = memop::GLOBAL_TO_SHARED;
	map[(int)InstrType::MemorySpace::SURFACE] = memop::SURFACE;
	map[(int)InstrType::MemorySpace::TEXTURE] = memop::TEXTURE;
	res.ms = map[(int)instr.getMemorySpace()];   
	res.s = instr.getSize();
	const char *op = instr.getOpcode();
	res.atomic = strncmp(op, "ATOMG", 5) == 0;
	return res;
}

uint32_t nctax, nctay, nctaz;
uint32_t ntx, nty, ntz;
std::size_t taboff;
uint64_t lmem_min_a = std::numeric_limits<uint64_t>::max();
uint64_t max_a = std::numeric_limits<uint64_t>::min();
void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
	/* Get related functions of the kernel (device function that can be
	 * called by the kernel) */
	std::vector<hipFunction_t> related_functions =
		nvbit_get_related_functions(ctx, func);

	/* add kernel itself to the related function vector */
	related_functions.push_back(func);

	/* iterate on function */
	for (auto f : related_functions) {
		/* "recording" function was instrumented, if set insertion failed
		 * we have already encountered this function */
		if (!already_instrumented.insert(f).second) {
			continue;
		}
		const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
		if (verbose) {
			printf("Inspecting function %s at address 0x%lx\n",
				   nvbit_get_func_name(ctx, f), nvbit_get_func_addr(f));
		}

		uint32_t cnt = 0;
		/* iterate on all the static instructions in the function */
		for (auto instr : instrs) {
			if (cnt < instr_begin_interval || cnt >= instr_end_interval ||
				instr->getMemorySpace() == InstrType::MemorySpace::NONE ||
				instr->getMemorySpace() == InstrType::MemorySpace::CONSTANT) {
				cnt++;
				continue;
			}

			if (opcode_to_id_map.find(instr->getOpcode()) ==
				opcode_to_id_map.end()) {
				uint32_t opcode_id = opcode_to_id_map.size();
				opcode_to_id_map[instr->getOpcode()] = opcode_id;
				const char *opstr = instr->getOpcode();
				uint32_t opstrlen = std::strlen(opstr);
				id_to_opcode_map[opcode_id] = std::string(opstr);
				memop mop = parse_op(*instr);
				memops[opcode_id] = mop;
			}

			int opcode_id = opcode_to_id_map[instr->getOpcode()];
			int mref_idx = 0;
			/* iterate on the operands */
			for (int i = 0; i < instr->getNumOperands(); i++) {
				/* get the operand "i" */
				const InstrType::operand_t* op = instr->getOperand(i);

				if (op->type == InstrType::OperandType::MREF) {
					/* insert call to the instrumentation function with its
					 * arguments */
					nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
					/* predicate value */
					nvbit_add_call_arg_guard_pred_val(instr);
					/* opcode id */
					nvbit_add_call_arg_const_val32(instr, opcode_id);
					/* memory reference 64 bit address */
					nvbit_add_call_arg_mref_addr64(instr, mref_idx);
					/* add pointer to channel_dev*/
					nvbit_add_call_arg_const_val64(instr,
												   (uint64_t)&channel_dev);
					mref_idx++;
				}
			}
			cnt++;
		}
//		 std::exit(1);
	}
	memop mop;
	mop.ld = mop.st = 0;
// 	os.write((const char*)&mop, sizeof(memop));

// 	std::cout << nctax<<nctay<<nctaz <<std::endl;
}

__global__ void flush_channel() {
	/* push memory access with negative cta id to communicate the kernel is
	 * completed */
	mem_access_t ma;
	ma.gwarpid = -1;
	channel_dev.push(&ma, sizeof(mem_access_t));

	/* flush channel */
	channel_dev.flush();
}

uint64_t translate_lmem_addr(uint64_t a, uint32_t gwarpid, int lane, int lmem_per_thread)
{
	uint64_t elm = a / 4;
	uint64_t off = a % 4;
	uint64_t translated = gwarpid * 32 * lmem_per_thread + (elm * 32 + lane) * 4 + off;
	return translated;
}
int lmem_static_nbytes;
struct MemAccList {
	std::list<mem_access_t> data;

	void clear(uint32_t x, uint32_t y, uint32_t z)
	{
		data.clear();
	}
	void operator()(const mem_access_t &ma)
	{
		data.push_back(ma);
	}
};



std::map<uint64_t, Alloc> allocs;
int allocidx = 0;
uint64_t allocidx2ptr[4096];
int kernelidx = 0;
void proc_memaccs(MemAccList &memaccblocks, int nwarps)
{
	cachesim.clear();

	cachesim.grid(nctax, nctay, nctaz, ntx, nty, ntz);

	for (void *handle : plugins) {
		decltype(&before_sim) cb = (decltype(&before_sim))dlsym(handle, "before_sim");
		(*cb)(cachesim);
	}

	std::list<mem_access_t> &allmemaccs = memaccblocks.data;

	std::cout << "Start proc" << std::endl;
	uint64_t local_min = std::numeric_limits<uint64_t>::max(), local_max = 0;

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	
	uint64_t lmem = lmem_static_nbytes * 32 * nwarps/* * deviceProp.multiProcessorCount*/;

	uint64_t num_raw = 0;
	uint64_t num_coa = 0;
	uint64_t num_raw_st = 0;
	uint64_t num_coa_st = 0;
	uint64_t num_local_raw = 0;
	uint64_t num_local_coa = 0;
	uint64_t num_local_raw_st = 0;
	uint64_t num_local_coa_st = 0;
	Alloc lmemcounter;
	for (mem_access_t &ma : allmemaccs) {
// 		uint32_t ncta = ma.cta_id_x + ma.cta_id_y * nctax + ma.cta_id_z * nctax * nctay;
		uint64_t addrs[32];
		int sizes[32];
		uint8_t footprints[32];
// 					hipDeviceptr_t addrs[32];
// 		nwarps = std::max(nwarps, (int)ma.warp_id);
		uint32_t origs[32], origoffs[33];
// 		if (!memops[ma.opcode_id].ld) continue; // TODO TODO REQUIRED FOR L2
		bool dead = true;
		for (int i = 0; i < 32; ++i) {
			if (!GETPRED(ma.preds, i)) continue;
			dead = false;
			break;
		}
		if (dead) continue;
		uint64_t srcaddrs[32];
		if (memops[ma.opcode_id].ms == memop::LOCAL) {
			for (int j = 0; j < memops[ma.opcode_id].s; j += 4) {
				int s = std::min(4, (int)memops[ma.opcode_id].s);
				int nn = 0; 
				for (int i = 0; i < 32; ++i) {
					if (!GETPRED(ma.preds, i)) continue;
					srcaddrs[i] = ma.a(i) - lmem_min_a;
					srcaddrs[i] = translate_lmem_addr(srcaddrs[i] + j, ma.gwarpid, i, lmem_static_nbytes);
					++nn;
				}
				int na = coalesce(srcaddrs, s, ma.preds, addrs, footprints, origoffs, origs);
				if (memops[ma.opcode_id].ld) {
					num_local_raw += 32;
					num_local_coa += na;
					++lmemcounter.ld_reqs;
					lmemcounter.ld_txn += na;
				}
				if (memops[ma.opcode_id].st) {
					num_local_raw_st += 32;
					num_local_coa_st += na;
					++lmemcounter.st_reqs;
					lmemcounter.st_txn += na;
				}
				int accs = memops[ma.opcode_id].ld + memops[ma.opcode_id].st;
				for (int k = 0; k < accs; ++k) {
					for (int i = 0; i < na; ++i) {
						uint64_t off = addrs[i];
						cachesim.lmem(off, footprints[i], ma.sm, i, k == 1, false /* write back *//*   k == 1 ? 2 : 0*/);
		// 							cachesim(off, sizes[i], cache->second.order, ma.sm);
					}
				}
			}
		} else {
			for (int i = 0; i < 32; ++i) {
				srcaddrs[i] = ma.a(i);
			}

			int na = coalesce(srcaddrs, memops[ma.opcode_id].s, ma.preds, addrs, footprints, origoffs, origs);
			if (!memops[ma.opcode_id].atomic) {
				if (memops[ma.opcode_id].ld) {
					num_raw += 32;
					num_coa += na;
				}
				if (memops[ma.opcode_id].st) {
					num_raw_st += 32;
					num_coa_st += na;
				}
			}
			int firstalloc = 0;
			bool first = true;
			for (int i = 0; i < na; ++i) {
				auto cache = allocs.empty() ? allocs.end() : std::prev(allocs.upper_bound(addrs[i]));
				if (cache == allocs.end() || addrs[i] < cache->first || addrs[i] + Clinesize > cache->first + (cache->second.size + Clinesize - 1) / Clinesize * Clinesize) {
					std::cout << "Seg fault 2 " << addrs[i] << std::endl;
					for (auto a : allocs) {
						std::cout << a.first << "+" << a.second.size << " " << a.second.order << std::endl;
					}
					std::exit(1);
				}
				if (!memops[ma.opcode_id].atomic) {
					if (memops[ma.opcode_id].ld) ++cache->second.ld_txn;
					if (memops[ma.opcode_id].st) ++cache->second.st_txn;
				}
				if (first) {
					first = false;
					if (!memops[ma.opcode_id].atomic) {
						if (memops[ma.opcode_id].ld) ++cache->second.ld_reqs;
						if (memops[ma.opcode_id].st) ++cache->second.st_reqs;
					}
					firstalloc = cache->second.order;
				} else {
					if (cache->second.order != firstalloc) {
						std::cout << "Different allocs accross warp?" << std::endl;
						std::exit(1);
					}
				}


				uint64_t off = addrs[i] - cache->first;

				uint8_t offs[32];
				int noffs = 0;
				for (int j = origoffs[i]; j < origoffs[i + 1]; ++j) {
					offs[noffs++] = srcaddrs[origs[j]] - addrs[i];
				}
				if (memops[ma.opcode_id].ld)
					cachesim(off, footprints[i], cache->second.order, ma.sm, origs + origoffs[i], offs, noffs, memops[ma.opcode_id].atomic, false, false, ma.gwarpid);
				if (memops[ma.opcode_id].st)
					cachesim(off, footprints[i], cache->second.order, ma.sm, origs + origoffs[i], offs, noffs, memops[ma.opcode_id].atomic, true, true /* write through */, ma.gwarpid);
			}
		}
	}
	allmemaccs = std::list<mem_access_t>();
// 	++nwarps;
	std::cout << "NWARPS: " << nwarps << std::endl;
	std::cout << "GLOBAL Coalesce efficiency (0 is best): " <<  (double)num_coa / num_raw << " #reqs: " << num_raw / 32 << " #transacs: " << num_coa << std::endl;
	std::cout << "GLOBAL Coalesce efficiency store (0 is best): " <<  (double)num_coa_st / num_raw_st << " #reqs: " << num_raw_st / 32 << " #transacs: " << num_coa_st << std::endl;
	std::cout << "LOCAL Coalesce efficiency (0 is best): " <<  (double)num_local_coa / num_local_raw << " #reqs: " << num_local_raw / 32 << " #transacs: " << num_local_coa << std::endl;
	std::cout << "LOCAL Coalesce efficiency store (0 is best): " <<  (double)num_local_coa_st / num_local_raw_st << " #reqs: " << num_local_raw_st / 32 << " #transacs: " << num_local_coa_st << std::endl;
	std::cout << "FINAL EXPN|" << num_raw / 32 << "|" << num_raw_st / 32 << "|" << num_coa << "|" << num_coa_st << "|" << num_local_raw / 32 << "|" << num_local_raw_st / 32 << "|" << num_local_coa << "|" << num_local_coa_st << std::endl;
	std::vector<uint32_t> allocvec(allocs.size());
	std::vector<Meminf> midescs(allocs.size(), Meminf{ _MI_MAX, -1 });
	std::unordered_map<uint64_t, uint32_t> ptr2buf;
	for (const auto &a : allocs) {
		std::cout << a.first << " " << a.second.size << std::endl;
		allocvec[a.second.order] = a.second.size;
		auto miit = meminfs.find(a.first);
		if (miit != meminfs.end()) {
			midescs[a.second.order] = miit->second;
		}
		ptr2buf[a.first] = a.second.order;
	}

	cachesim.register_allocs(allocs, lmemcounter);
	cachesim.process(deviceProp.multiProcessorCount, nwarps, lmem_static_nbytes/*, allocidx2ptr, midescs.data(), allocvec.data(), allocs.size()*/);

	for (void *handle : plugins) {
		decltype(&after_sim) cb = (decltype(&after_sim))dlsym(handle, "after_sim");
// 		(*cb)(cachesim);
	}
}


// std::list<mem_access_t> allmemaccs;
MemAccList allmemaccs;
int nwarps = -1;
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
						 const char *name, void *params, hipError_t *pStatus) {
	if (skip_flag) return;

	if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
		cbid == API_CUDA_cuLaunchKernel) {
		cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

		if (!is_exit) {
			int nregs;
			CUDA_SAFECALL(
				hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

			int shmem_static_nbytes;
			CUDA_SAFECALL(
				hipFuncGetAttribute(&shmem_static_nbytes,
								   HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));


			instrument_function_if_needed(ctx, p->f);

			nvbit_enable_instrumented(ctx, p->f, true);

			nctax = p->gridDimX;
			nctay = p->gridDimY;
			nctaz = p->gridDimZ;
			ntx = p->blockDimX;
			nty = p->blockDimY;
			ntz = p->blockDimZ;
			CUDA_SAFECALL(
				hipFuncGetAttribute(&lmem_static_nbytes,
									HIP_FUNC_ATTRIBUTE_LOCAL_SIZE_BYTES, p->f));


			uint64_t lmem_base = nvbit_get_local_mem_base_addr(ctx);
			std::cout << "LMEM: " << lmem_base << " +" << lmem_static_nbytes << " per thread" << std::endl;
// 			uint32_t _vals[] = { nctax, nctay, nctaz, p->blockDimX, p->blockDimY, p->blockDimZ, nregs, shmem_static_nbytes + p->sharedMemBytes, lmem_static_nbytes };


			printf(
				"Kernel %s - grid size %d,%d,%d - block size %d,%d,%d - nregs "
				"%d - shmem %d - cuda stream id %ld\n",
				nvbit_get_func_name(ctx, p->f), p->gridDimX, p->gridDimY,
				p->gridDimZ, p->blockDimX, p->blockDimY, p->blockDimZ, nregs,
				shmem_static_nbytes + p->sharedMemBytes, (uint64_t)p->hStream);
			allmemaccs.clear(nctax, nctay, nctaz);
			allocs.clear(); allocidx = 0;
// 			nwarps = -1;
			uint32_t nb = nctax * nctay * nctaz;
			uint32_t nt = p->blockDimZ * p->blockDimX * p->blockDimY;
			uint32_t nw = (nt + 31) / 32;
			nwarps = nb * nw;
			recv_thread_receiving = true;
			lmem_min_a = std::numeric_limits<uint64_t>::max();

		} else {
			/* make sure current kernel is completed */
			hipDeviceSynchronize();
			assert(hipGetLastError() == hipSuccess);

			/* make sure we prevent re-entry on the nvbit_callback when issuing
			 * the flush_channel kernel */
			skip_flag = true;

			/* issue flush of channel so we are sure all the memory accesses
			 * have been pushed */
			flush_channel<<<1, 1>>>();
			hipDeviceSynchronize();
			assert(hipGetLastError() == hipSuccess);

			/* unset the skip flag */
			skip_flag = false;

			/* wait here until the receiving thread has not finished with the
			 * current kernel */
			while (recv_thread_receiving) {
				pthread_yield();
			}

			proc_memaccs(allmemaccs, nwarps);
		}
	}
}


void *recv_thread_fun(void *) 
{
	char *recv_buffer = (char *)malloc(CHANNEL_SIZE);
	while (recv_thread_started) {
		uint32_t num_recv_bytes = 0;
		if (recv_thread_receiving &&
			(num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) >
				0) {
			uint32_t num_processed_bytes = 0;
			while (num_processed_bytes < num_recv_bytes) {
				mem_access_t *ma =
					(mem_access_t *)&recv_buffer[num_processed_bytes];

				/* when we get this cta_id_x it means the kernel has completed
				 */
				if (ma->gwarpid == (uint64_t)-1) {
					recv_thread_receiving = false;
					break;
				}

				if (ma->opcode_id != 0xfffffffeu && (memops[ma->opcode_id].ms == memop::LOCAL || memops[ma->opcode_id].ms == memop::GLOBAL)) {
					if (memops[ma->opcode_id].ms == memop::GLOBAL) {
						for (int i = 0; i < 32; ++i) {
							if (!GETPRED(ma->preds, i)) continue;
							auto cache = allocs.empty() ? allocs.end() : std::prev(allocs.upper_bound(ma->a(i)));
							if (cache == allocs.end() || ma->a(i) < cache->first || ma->a(i) + memops[ma->opcode_id].s > cache->first + (cache->second.size + Clinesize - 1) / Clinesize * Clinesize) {
								hipDeviceptr_t p, s;
								if (hipPointerGetAttribute(&p, HIP_POINTER_ATTRIBUTE_RANGE_START_ADDR, ma->a(i)) == hipSuccess && hipPointerGetAttribute(&s, HIP_POINTER_ATTRIBUTE_RANGE_SIZE, ma->a(i)) == hipSuccess) {
									cache = allocs.find(p);
									if (cache != allocs.end()) { // this wont happen due to clear()
										std::cout << "AGAIN " << p << "+" << s << " @" << cache->second.order << std::endl;
										cache->second.size = s;
									} else {
										if (cache != allocs.end()) std::cout << cache->first << std::endl;
										std::cout << "ADDING " << p << "+" << s << " @" << allocidx << std::endl;
										cache = allocs.emplace(p, Alloc{ s, allocidx++, 0/*shm*/, 0, 0 }).first;
									}
								} else {
									std::cerr << "SEG FAULT " << ma->a(i) << " " << memops[ma->opcode_id].ms << std::endl;
									std::exit(1);
								}
							} else if (cache != allocs.end()) {
							}
						}
					} else if (memops[ma->opcode_id].ms == memop::LOCAL) {
						for (int i = 0; i < 32; ++i) {
							if (!GETPRED(ma->preds, i)) continue;
							lmem_min_a = std::min(lmem_min_a, ma->a(i));
						}
					}
					allmemaccs(*ma);
				}

				num_processed_bytes += sizeof(mem_access_t);
			}
		}
	}

	free(recv_buffer);


	return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx)
{
	recv_thread_started = true;
	channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
	pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);
}

void nvbit_at_ctx_term(hipCtx_t ctx)
{
	if (recv_thread_started) {
		recv_thread_started = false;
		pthread_join(recv_thread, NULL);
	}
	for (void *handle : plugins) {
		dlclose(handle);
	}
}
